#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include "Maps/MapsManager.h"

#include "bilLib/Las1_3_handler.h"
#include "MarchingCubes/MarchingCubes.h"

__global__ // This keyword means the code runs on the GPU.
void add(int n, float *x, float *y)
    {
    printf("Hello World!\n");
    // At each index, add x to y.
    for (int i = 0; i < n; i++)
        {
        y[i] = x[i] + y[i];
        }
    }

int main(void)
    {
    int N = 100;
    float *x, *y;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    // Initialize our x and y arrays with some floats.
    for (int i = 0; i < N; i++)
        {
        x[i] = 1.0f;
        y[i] = 2.0f;
        }

    // Run the function on using the GPU.
    add<<<1, 1>>>(N, x, y); // Notice the brackets.

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        {
        maxError = fmax(maxError, fabs(y[i]-3.0f));
        }
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;
    }
